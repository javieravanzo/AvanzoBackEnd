#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include "omp.h"
#include <iostream>


using namespace std;
using namespace cv;

//nvcc blur-effect.cu -o blur-effect `pkg-config opencv --libs`
int rows;
int cols;
int** img;

using namespace cv;
struct pixel{
  int R, G, B;
};

char add_c[9] = {'m','o','d','i','f','i','e','d','_'};

__global__ void blur(pixel *image, int KERNEL_SIZE, int totalThreads, int rows, int cols){
  int initIteration, endIteration;
  int i,j,k,l, index;
  index = (blockDim.x * blockIdx.x) + threadIdx.x;
  initIteration = (rows/totalThreads)*index;
  endIteration = initIteration + (rows/totalThreads) - 1;
  if(index == totalThreads -1 ){
    endIteration = rows-1;
  }

  printf("start : %d   end: %d", initIteration, endIteration);

  for(i = initIteration; i <= endIteration; i++){
    for(j = 0; j < cols; j++){

      int x1, x2;
      int y1, y2;

      //Verifica y asigna la cantidad de espacios para hacer el promedio
      if( j - KERNEL_SIZE >= 0 )
      x1 = j - KERNEL_SIZE;
      else
      x1 = 0;

      if( j + KERNEL_SIZE < cols )
      x2 = j + KERNEL_SIZE;
      else
      x2 = cols-1;

      if( i - KERNEL_SIZE >= 0 )
      y1 = i - KERNEL_SIZE;
      else
      y1 = 0;

      if( i + KERNEL_SIZE < rows )
      y2 = i + KERNEL_SIZE;
      else
      y2 = rows-1;


      int b = 0, r = 0, g = 0, c = 0;

      for( k = y1; k <= y2; k++){
        for( l = x1; l <= x2; l++ ){
          b += image[k*cols +l].B;
          g += image[k*cols +l].G;
          r += image[k*cols +l].R;
          c++;
        }
      }


      image[i*cols + j].B = (b/c);
      image[i*cols + j].G = (g/c);
      image[i*cols + j].R = (r/c);



    }
  }



}

int* average( int row, int column, int kernel){

  int x1, x2;
  int y1, y2;

  //Verifica y asigna la cantidad de espacios para hacer el promedio
  if( column - kernel >= 0 )
  x1 = column - kernel;
  else
  x1 = 0;

  if( column + kernel < cols )
  x2 = column + kernel;
  else
  x2 = cols-1;

  if( row - kernel >= 0 )
  y1 = row - kernel;
  else
  y1 = 0;

  if( row + kernel < rows )
  y2 = row + kernel;
  else
  y2 = rows-1;

  int R = 0, G = 0, B = 0;


  //ALmacena los valores de R, G y B  en cada capa del pixel.
  for( int i = y1; i <= y2; i++){
    for( int j = x1; j <= x2; j++ ){
      int * temp = *(img + i * cols + j);
      R += *(temp + 2);
      G += *(temp + 1);
      B += *(temp);
    }
  }

  //Saca el promedio para cada pixel diviendo entre el tamaño de kernel.
  int* ans = new int[3];
  int size = (x2 - x1 + 1)*(y2 - y1 + 1);
  ans[0] = (int)(B / size);
  ans[1] = (int)(G / size);
  ans[2] = (int)(R / size);
  return ans;

}


int main(int argc, const char *argv[]){
  int N_THREADS;
  int N_BLOCKS;
  int KERNEL_SIZE;
  int flag = 0;
  hipError_t err = hipSuccess;



  if( argc < 3){
    printf("Usage: <img_path> <kernel_size> <thread_number> \n");
    return -1;
  }
  sscanf(argv[2],"%d", &KERNEL_SIZE);
  sscanf(argv[3],"%d", &N_BLOCKS);
  sscanf(argv[4],"%d", &N_THREADS);

  if(N_THREADS == 1){
    flag=1;
  }

  Mat image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

  int size = image.rows*image.cols*sizeof(struct pixel);

  pixel * im = (pixel*)malloc(size);

  char * name = (char*)malloc(sizeof(argv[1]) + sizeof(add_c) + 1);
  name[0] = 0;
  strcat(name,add_c);
  strcat(name,argv[1]);

  if(flag==1){

    char* nameImage = name;


    //Verificar el contenido de la imagen
    if( !image.data ){
      cout << "No image data" << endl;
      return -1;
    }

    //Identifica la cantidad de filas y columnas de la imagen escrita como parámetro.
    rows = image.rows;
    cols = image.cols;
    img = new int * [rows * cols];


    //Lee el argumento del kernel.
    int kernel = atoi( argv[2] );

    omp_set_num_threads(flag);
    #pragma omp parallel for
    for( int i = 0; i < rows * cols; i++) {
      *(img + i) = new int[3];
      *(*(img + i))  = image.at<Vec3b>( int(i / cols), i % cols )[0];
      *(*(img + i) + 1) = image.at<Vec3b>( int(i / cols), i % cols )[1];
      *(*(img + i) + 2) = image.at<Vec3b>( int(i / cols), i % cols )[2];
    }

    //Asigna el nuevo pixel (promediado) a la imagen que se tenía.
    omp_set_num_threads(flag);
    #pragma omp parallel for
    for( int r = 0; r < rows; r++ ){
      for( int c = 0; c < cols; c++ ){
        int* BGR = average( r, c, int((kernel - 1)/0.5) );
        image.at<Vec3b>( r, c )[2] = BGR[2];
        image.at<Vec3b>( r, c )[1] = BGR[1];
        image.at<Vec3b>( r, c )[0] = BGR[0];
      }
    }
    imwrite( nameImage, image);
  }


  if(flag ==0){


    for(int i = 0; i < image.rows; i++){
      for(int j = 0; j < image.cols; j++){
        im[i*image.cols +j].B = image.at<Vec3b>(i,j)[0];
        im[i*image.cols +j].G = image.at<Vec3b>(i,j)[1];
        im[i*image.cols+ j].R = image.at<Vec3b>(i,j)[2];
      }
    }

    pixel * d_im;
    err = hipMalloc((void**)&d_im, sizeof(struct pixel)*image.rows*image.cols);
    if (err != hipSuccess){
      fprintf(stderr, "(error allocating in device code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    printf("allocated\n");

    err = hipMemcpy(d_im, im, size, hipMemcpyHostToDevice);

    if (err != hipSuccess){
      fprintf(stderr, "error copy from host to device( code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    printf("copy from host to device\n");


    printf("launching kernel\n");
    blur<<<N_BLOCKS, N_THREADS>>>(d_im, KERNEL_SIZE, N_THREADS, image.rows, image.cols);

    err = hipGetLastError();
    if (err != hipSuccess){
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    printf("kernel finished\n");

    err = hipMemcpy(im, d_im, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess){
      fprintf(stderr, "error copy from device to host( code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    printf("copy from device to host\n");

    for(int i = 0; i < image.rows; i++){
      for(int j = 0; j < image.cols; j++){
        image.at<Vec3b>(i,j)[0] = im[i*image.cols + j].B;
        image.at<Vec3b>(i,j)[1] = im[i*image.cols + j].G;
        image.at<Vec3b>(i,j)[2] = im[i*image.cols + j].R;
      }
    }
    if(!image.data){
      printf("no image data\n");
      return -1;
    }

    imwrite(name, image);

    free(im);
    err = hipFree(d_im);

    if (err != hipSuccess){
      fprintf(stderr, "error free( code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }
  return 0;
}
